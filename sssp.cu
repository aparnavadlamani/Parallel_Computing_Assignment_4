#include "hip/hip_runtime.h"

#include<iostream>
#include<limits.h>

#define NUM_NODES 5

using namespace std;

__global__ void CUDA_SSSP_KERNEL1(int *Va, int *Ea, int *Wa, bool *Ma, int *Ca, int *Ua, bool *done)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id>NUM_NODES)
	{
		*done = false;
	}
	if(Ma[id]==true)
	{
		Ma[id] = false;
		__syncthreads(); 
		int start = Va[id];
		int end = Va[id+1];
		for(int i=start;i<end;i++) 
		{
			int nid = Ea[i];

			if(Ua[nid]>(Ca[nid]+Wa[nid]))
			{
				Ua[nid] = Ca[id] + Wa[nid];
			}
		}
	}
}
__global__ void CUDA_SSSP_KERNEL2(int *Va, int *Ea, int *Wa, bool *Ma, int *Ca, int *Ua, bool *done)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id>NUM_NODES)
	{
		*done = false;
	}
	if(Ca[id]>Ua[id])
	{
		Ca[id] = Ua[id];
		Ma[id] = true;
		*done = false;
	}
	Ua[id] = Ca[id];
}

int main(int argc, char** argv)
{
	int** graph = new int* [NUM_NODES];
	int edges = 0;
    for(int i=0;i<NUM_NODES;i++)
    {
        graph[i] = new int[NUM_NODES];
    }
    for(int i=0;i<NUM_NODES;i++)
    {
        for(int j=i+1;j<NUM_NODES;j++)
        {
			int x = rand()%100;
            if(x!=0)
            {
				edges+=1;
                graph[i][j] = x;
                graph[j][i] = x;
            }
        }
    }
	
	int* v = new int[NUM_NODES+1];
	int* e = new int[2*edges];
	int* w = new int[2*edges];
	int x = 0;
	for(int i=0;i<NUM_NODES;i++)
	{
		v[i] = x;
		for(int j=0;j<NUM_NODES;j++)
		{
			if(graph[i][j]!=0)
			{
				e[x] = j;
				w[x] = graph[i][j];
				x+=1;
			}
		}
	}
	v[NUM_NODES] = x;
	bool mask[NUM_NODES] = { false };
	int cost[NUM_NODES] = { INT_MAX };
	int updated[NUM_NODES] = { INT_MAX };

	int source = 0;
	mask[source] = true;
	updated[source] = 0;
	cost[source] = 0;

	int* Va;
	hipMalloc(&Va, sizeof(int)*(NUM_NODES+1));
	hipMemcpy(Va, v, sizeof(int)*(NUM_NODES+1), hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc(&Ea, sizeof(int)*(2*edges));
	hipMemcpy(Ea, e, sizeof(int)*(2*edges), hipMemcpyHostToDevice);

	int* Wa;
	hipMalloc(&Wa, sizeof(int)*(2*edges));
	hipMemcpy(Wa, w, sizeof(int)*(2*edges), hipMemcpyHostToDevice);

	bool* Ma;
	hipMalloc(&Ma, sizeof(bool)*NUM_NODES);
	hipMemcpy(Ma, mask, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ua;
	hipMalloc(&Ua, sizeof(int)*NUM_NODES);
	hipMemcpy(Ua, updated, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc(&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	int num_blks = 1;
	int threads = 5;

	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	
	do {
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_SSSP_KERNEL1<<<num_blks, threads>>>(Va, Ea, Wa, Ma, Ca, Ua, d_done);
		CUDA_SSSP_KERNEL2<<<num_blks, threads>>>(Va, Ea, Wa, Ma, Ca, Ua, d_done);
		hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost);
	} while (!done);

	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);
	
	cout<<"Cost: "<<endl;
	for(int i=0;i<NUM_NODES;i++)
	{
		cout<<cost[i]<<" ";
	}
	cout<<endl;
	return 0;
}

